#include "hip/hip_runtime.h"
#include <windows.h>
#include <winbase.h>
//#include "stdafx.h"
#include <stdio.h>
#include <kernelspecs.h>
#include <stdlib.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <io.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include ""
#include <DbgEng.h>
#include <process.h>
#include <psapi.h>
#include <time.h>

#include <winsock.h>
/**/



__global__ void copyKeyboardBuffer(PCHAR inBuffer, PCHAR outBuffer) {
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	outBuffer[index] = inBuffer[index];
}

__global__ void sendTextToGpu(PCHAR keyboardText) {
	printf("In GPU\n");
	printf("GPU: [0x%lx]\n", keyboardText);
	printf("GPU: %s\n", keyboardText);
}

int main(int argc, char * argv[]) {


	PCHAR textForGpu = (PCHAR)malloc(sizeof(CHAR) * 2048);


	PCHAR d_OutgoingKeystrokeBuffer;
	PCHAR h_OutgoingKeystrokeBuffer;


	h_OutgoingKeystrokeBuffer = (PCHAR)malloc(sizeof(CHAR) * 2048);



	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));
	checkCudaErrors(hipHostRegister(h_OutgoingKeystrokeBuffer, sizeof(CHAR) * 2048, hipHostRegisterMapped)); // zzy
	checkCudaErrors(hipHostGetDevicePointer((void **)&d_OutgoingKeystrokeBuffer, (void *)h_OutgoingKeystrokeBuffer, 0)); // zzy

	while (TRUE) {
		printf("enter text:");
		gets_s(textForGpu, 2048);
		size_t textLength = strlen(textForGpu);

		printf("got %d characters\n", textLength);
		if (textLength == 0) {
			break;
		}
		else {
			textLength++;
			PCHAR d_Text;
			checkCudaErrors(hipMalloc((void **)&d_Text, sizeof(char) * textLength));
			checkCudaErrors(hipMemcpy(d_Text, textForGpu, textLength * sizeof(char), hipMemcpyHostToDevice));
			printf("Launching CUDA process.\n");

			
			copyKeyboardBuffer<<<4, 512 >>>(d_Text, d_OutgoingKeystrokeBuffer);
			checkCudaErrors(hipDeviceSynchronize());

			//printf("is this the right string? %s\n", h_OutgoingKeystrokeBuffer);
			sendTextToGpu <<<1, 1 >>>(d_Text);
			checkCudaErrors(hipDeviceSynchronize());
		}
	}
	return 0;
}




